#include <cstdio>
#include <hip/hip_runtime.h>

constexpr int IN  = 4;
constexpr int HID = 5;
constexpr int OUT = 1;
constexpr int BATCH = 1<<20;          // 1 M samples for timing

__global__ void hidden(const float* W, const float* b,
                       const float* x, float* h){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= BATCH * HID) return;
    int sample = idx / HID, neuron = idx % HID;
    float s = b[neuron];
    #pragma unroll
    for(int i=0;i<IN;++i)
        s += W[neuron*IN+i] * x[sample*IN+i];
    h[idx] = s > 0.f ? s : 0.f;       // ReLU
}
__global__ void output(const float* W, const float* b,
                       const float* h, float* y){
    int sample = blockIdx.x * blockDim.x + threadIdx.x;
    if(sample >= BATCH) return;
    float s = b[0];
    #pragma unroll
    for(int i=0;i<HID;++i)
        s += W[i] * h[sample*HID+i];
    y[sample] = s;                    // linear output
}

int main(){
    size_t xBytes = BATCH*IN*sizeof(float);
    size_t hBytes = BATCH*HID*sizeof(float);

    float *dx,*dWh,*dbh,*dWo,*dbo,*dh,*dy;
    hipMalloc(&dx,xBytes);   hipMalloc(&dh,hBytes);  hipMalloc(&dy,BATCH*sizeof(float));
    hipMalloc(&dWh,HID*IN*sizeof(float));
    hipMalloc(&dbh,HID*sizeof(float));
    hipMalloc(&dWo,HID*sizeof(float));
    hipMalloc(&dbo,OUT*sizeof(float));

    // fill with junk data once (random on host, memcpy is fine)
    float *dummy = (float*)malloc(hBytes> xBytes? hBytes: xBytes);
    hipMemcpy(dx, dummy, xBytes, hipMemcpyHostToDevice);
    hipMemcpy(dWh,dummy,HID*IN*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dbh,dummy,HID*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dWo,dummy,HID*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dbo,dummy,OUT*sizeof(float),hipMemcpyHostToDevice);

    dim3 block(256);
    hipEvent_t t0,t1; hipEventCreate(&t0); hipEventCreate(&t1);
    hipEventRecord(t0);
    hidden<<<(BATCH*HID+255)/256,block>>>(dWh,dbh,dx,dh);
    output<<<(BATCH+255)/256,block>>>(dWo,dbo,dh,dy);
    hipEventRecord(t1); hipEventSynchronize(t1);
    float ms=0; hipEventElapsedTime(&ms,t0,t1);
    printf("CUDA forward pass: %.3f ms\\n",ms);
    return 0;
}
