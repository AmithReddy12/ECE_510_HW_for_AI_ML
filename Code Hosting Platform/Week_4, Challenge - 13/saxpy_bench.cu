/**
 * saxpy_bench.cu  -- CUDA SAXPY micro‑benchmark
 * Measures total vs kernel‑only runtime for N = 2^15 … 2^25.
 * Compile: nvcc -O3 saxpy_bench.cu -o saxpy_bench
 * Run:     ./saxpy_bench > saxpy_times.csv
 */
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float a, float* x, float* y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main()
{
    const int first_pow = 15, last_pow = 25;            // 2^15 … 2^25
    for (int p = first_pow; p <= last_pow; ++p)
    {
        int    N     = 1 << p;
        size_t bytes = N * sizeof(float);

        // host buffers
        float *h_x = (float*)malloc(bytes);
        float *h_y = (float*)malloc(bytes);
        for (int i = 0; i < N; ++i) { h_x[i] = 1.0f; h_y[i] = 2.0f; }

        // device buffers
        float *d_x, *d_y;
        hipMalloc(&d_x, bytes);
        hipMalloc(&d_y, bytes);

        hipMemcpy(d_x, h_x, bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_y, h_y, bytes, hipMemcpyHostToDevice);

        dim3 block(256);
        dim3 grid((N + block.x - 1) / block.x);

        hipEvent_t t0, t1;
        hipEventCreate(&t0);  hipEventCreate(&t1);

        // --- TOTAL time (H2D + kernel + D2H) -------------------------
        hipEventRecord(t0);
        saxpy<<<grid, block>>>(N, 2.0f, d_x, d_y);
        hipMemcpy(h_y, d_y, bytes, hipMemcpyDeviceToHost);
        hipEventRecord(t1);  hipEventSynchronize(t1);
        float total_ms = 0.0f;
        hipEventElapsedTime(&total_ms, t0, t1);

        // --- KERNEL‑ONLY time ---------------------------------------
        hipEventRecord(t0);
        saxpy<<<grid, block>>>(N, 2.0f, d_x, d_y);
        hipEventRecord(t1);  hipEventSynchronize(t1);
        float kernel_ms = 0.0f;
        hipEventElapsedTime(&kernel_ms, t0, t1);

        printf("%d,%.6f,%.6f\n", N, total_ms, kernel_ms);   // CSV line

        hipFree(d_x); hipFree(d_y); free(h_x); free(h_y);
        hipEventDestroy(t0); hipEventDestroy(t1);
    }
    return 0;
}
